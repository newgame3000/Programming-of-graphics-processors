
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <string>

using namespace std;

#define CSC(call) \
do { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        fprintf(stderr, "ERROR is %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status)); \
        exit(0); \
    } \
} while(0)


__device__ double conv(double r, double g, double b) {
    return 0.299 * r + 0.587 * g + 0.114 * b;
}

__global__ void kernel(hipTextureObject_t texObj, uchar4 *out, int w, int h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;

    double mx[3][3] = {
        {-1, 0, 1},
        {-1, 0, 1},
        {-1, 0, 1}
    };

    double my[3][3] = {
        {-1, -1, -1},
        {0, 0, 0},
        {1, 1, 1}
    };

    for(y = idy; y < h; y += offsety) {
        for(x = idx; x < w; x += offsetx) {
            
            double gx = 0;
            double gy = 0;

            for (int i = -1; i < 2; ++i) {
                for (int j = -1; j < 2; ++j) {

                    p = tex2D<uchar4>(texObj, x + i, y + j);

                    gx += mx[i + 1][j + 1] * conv(p.x, p.y, p.z);
                    gy += my[i + 1][j + 1] * conv(p.x, p.y, p.z);

                }
            }

            int g = min(255, int(sqrt(gx * gx + gy * gy)));

            out[y * w + x] = make_uchar4(g, g, g, p.w);
        }
    }
}

int main() {

    string in_name, out_name;
    cin >> in_name >> out_name;
    int w, h;

    FILE *in;

    if ((in = fopen(in_name.c_str(), "rb")) == NULL) {
        cout << "File open error\n";
        return -1;
    }

    fread(&w, sizeof(int), 1, in);
    fread(&h, sizeof(int), 1, in);
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, in);


    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = false;

    hipTextureObject_t texObj = 0;
    CSC(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));


    uchar4 *dev_out;
    CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));



    kernel<<< dim3(16, 16), dim3(32, 32) >>>(texObj, dev_out, w, h);
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    FILE *out;

    if ((out = fopen(out_name.c_str(), "wb")) == NULL) {
        cout << "File open error\n";
        return -1;
    }

    fwrite(&w, sizeof(int), 1, out);
    fwrite(&h, sizeof(int), 1, out);
    fwrite(data, sizeof(uchar4), w * h, out);


    fclose(in);
    fclose(out);

    hipDestroyTextureObject(texObj);
    hipFreeArray(arr);
    free(data);
    return 0;
}